#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <hip/hip_runtime.h>
#include <cmath>

// Kernel loop macro
#define CUDA_1D_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

template <typename T>
__device__ T bilinear_interpolate(
    const T* bottom_data, const int height, const int width,
    T y, T x, const int index) {
  if (y < -1.0 || y > height || x < -1.0 || x > width) return 0;

  y = max(y, T(0));
  x = max(x, T(0));

  int y_low = (int)y;
  int x_low = (int)x;
  int y_high = min(y_low + 1, height - 1);
  int x_high = min(x_low + 1, width - 1);

  T ly = y - y_low, lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;

  T v1 = bottom_data[y_low * width + x_low];
  T v2 = bottom_data[y_low * width + x_high];
  T v3 = bottom_data[y_high * width + x_low];
  T v4 = bottom_data[y_high * width + x_high];

  return (hy * hx * v1 + hy * lx * v2 + ly * hx * v3 + ly * lx * v4);
}

template <typename T>
__global__ void RoIAlignForward(
    const int nthreads, const T* bottom_data, const T spatial_scale,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width,
    const int sampling_ratio, const T* bottom_rois, T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    T roi_start_w = offset_bottom_rois[1] * spatial_scale;
    T roi_start_h = offset_bottom_rois[2] * spatial_scale;
    T roi_end_w = offset_bottom_rois[3] * spatial_scale;
    T roi_end_h = offset_bottom_rois[4] * spatial_scale;

    T roi_width = max(roi_end_w - roi_start_w, (T)1.);
    T roi_height = max(roi_end_h - roi_start_h, (T)1.);
    T bin_size_h = roi_height / static_cast<T>(pooled_height);
    T bin_size_w = roi_width / static_cast<T>(pooled_width);

    const T* offset_bottom_data =
        bottom_data + (roi_batch_ind * channels + c) * height * width;

    int roi_bin_grid_h = sampling_ratio > 0
                             ? sampling_ratio
                             : ceil(roi_height / pooled_height);
    int roi_bin_grid_w = sampling_ratio > 0
                             ? sampling_ratio
                             : ceil(roi_width / pooled_width);
    const T count = max(roi_bin_grid_h * roi_bin_grid_w, 1);

    T output_val = 0.;
    for (int iy = 0; iy < roi_bin_grid_h; iy++) {
      T y = roi_start_h + ph * bin_size_h + (iy + 0.5f) * bin_size_h / roi_bin_grid_h;
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        T x = roi_start_w + pw * bin_size_w + (ix + 0.5f) * bin_size_w / roi_bin_grid_w;
        output_val += bilinear_interpolate(offset_bottom_data, height, width, y, x, index);
      }
    }
    top_data[index] = output_val / count;
  }
}

template <typename T>
__device__ void bilinear_interpolate_gradient(
    const int height, const int width, T y, T x,
    T& w1, T& w2, T& w3, T& w4,
    int& x_low, int& x_high, int& y_low, int& y_high, const int index) {
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    w1 = w2 = w3 = w4 = 0.;
    x_low = x_high = y_low = y_high = -1;
    return;
  }

  y = max(y, T(0));
  x = max(x, T(0));

  y_low = (int)y;
  x_low = (int)x;
  y_high = min(y_low + 1, height - 1);
  x_high = min(x_low + 1, width - 1);

  T ly = y - y_low, lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;

  w1 = hy * hx;
  w2 = hy * lx;
  w3 = ly * hx;
  w4 = ly * lx;
}

template <typename T>
__global__ void RoIAlignBackwardFeature(
    const int nthreads, const T* top_diff, const int num_rois,
    const T spatial_scale, const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width,
    const int sampling_ratio, T* bottom_diff, const T* bottom_rois) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    T roi_start_w = offset_bottom_rois[1] * spatial_scale;
    T roi_start_h = offset_bottom_rois[2] * spatial_scale;
    T roi_end_w = offset_bottom_rois[3] * spatial_scale;
    T roi_end_h = offset_bottom_rois[4] * spatial_scale;

    T roi_width = max(roi_end_w - roi_start_w, (T)1.);
    T roi_height = max(roi_end_h - roi_start_h, (T)1.);
    T bin_size_h = roi_height / static_cast<T>(pooled_height);
    T bin_size_w = roi_width / static_cast<T>(pooled_width);

    T* offset_bottom_diff =
        bottom_diff + (roi_batch_ind * channels + c) * height * width;

    int top_offset = (n * channels + c) * pooled_height * pooled_width;
    const T* offset_top_diff = top_diff + top_offset;
    const T top_diff_this_bin = offset_top_diff[ph * pooled_width + pw];

    int roi_bin_grid_h = sampling_ratio > 0
                             ? sampling_ratio
                             : ceil(roi_height / pooled_height);
    int roi_bin_grid_w = sampling_ratio > 0
                             ? sampling_ratio
                             : ceil(roi_width / pooled_width);
    const T count = max(roi_bin_grid_h * roi_bin_grid_w, 1);

    for (int iy = 0; iy < roi_bin_grid_h; iy++) {
      T y = roi_start_h + ph * bin_size_h + (iy + 0.5f) * bin_size_h / roi_bin_grid_h;
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        T x = roi_start_w + pw * bin_size_w + (ix + 0.5f) * bin_size_w / roi_bin_grid_w;

        T w1, w2, w3, w4;
        int x_low, x_high, y_low, y_high;
        bilinear_interpolate_gradient(height, width, y, x, w1, w2, w3, w4,
                                      x_low, x_high, y_low, y_high, index);

        T g1 = top_diff_this_bin * w1 / count;
        T g2 = top_diff_this_bin * w2 / count;
        T g3 = top_diff_this_bin * w3 / count;
        T g4 = top_diff_this_bin * w4 / count;

        if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0) {
          atomicAdd(offset_bottom_diff + y_low * width + x_low, g1);
          atomicAdd(offset_bottom_diff + y_low * width + x_high, g2);
          atomicAdd(offset_bottom_diff + y_high * width + x_low, g3);
          atomicAdd(offset_bottom_diff + y_high * width + x_high, g4);
        }
      }
    }
  }
}

// Host interfaces

at::Tensor ROIAlign_forward_cuda(
    const at::Tensor& input, const at::Tensor& rois, float spatial_scale,
    int pooled_height, int pooled_width, int sampling_ratio) {
  TORCH_CHECK(input.is_cuda(), "input must be a CUDA tensor");
  TORCH_CHECK(rois.is_cuda(), "rois must be a CUDA tensor");

  auto num_rois = rois.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  auto output = at::zeros({num_rois, channels, pooled_height, pooled_width}, input.options());
  int output_size = num_rois * channels * pooled_height * pooled_width;

  const int threads = 512;
  const int blocks = (output_size + threads - 1) / threads;

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "ROIAlign_forward_cuda", [&] {
    RoIAlignForward<scalar_t><<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
        output_size,
        input.contiguous().data_ptr<scalar_t>(),
        spatial_scale,
        channels,
        height,
        width,
        pooled_height,
        pooled_width,
        sampling_ratio,
        rois.contiguous().data_ptr<scalar_t>(),
        output.data_ptr<scalar_t>());
  });

  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return output;
}

at::Tensor ROIAlign_backward_cuda(
    const at::Tensor& grad, const at::Tensor& rois,
    float spatial_scale, int pooled_height, int pooled_width,
    int batch_size, int channels, int height, int width,
    int sampling_ratio) {
  TORCH_CHECK(grad.is_cuda(), "grad must be a CUDA tensor");
  TORCH_CHECK(rois.is_cuda(), "rois must be a CUDA tensor");

  auto grad_input = at::zeros({batch_size, channels, height, width}, grad.options());
  int grad_size = grad.numel();

  const int threads = 512;
  const int blocks = (grad_size + threads - 1) / threads;

  AT_DISPATCH_FLOATING_TYPES(grad.scalar_type(), "ROIAlign_backward_cuda", [&] {
    RoIAlignBackwardFeature<scalar_t><<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
        grad_size,
        grad.contiguous().data_ptr<scalar_t>(),
        rois.size(0),
        spatial_scale,
        channels,
        height,
        width,
        pooled_height,
        pooled_width,
        sampling_ratio,
        grad_input.data_ptr<scalar_t>(),
        rois.contiguous().data_ptr<scalar_t>());
  });

  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return grad_input;
}
